
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

const int DSIZE = 40960;
const int block_size = 256;
const int grid_size = DSIZE/block_size;


__global__ void vector_addition(float* h_A, float* h_B) {

    //FIXME:
    // Express the vector index in terms of threads and blocks
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Swap the vector elements - make sure you are not out of range
    if (idx < DSIZE) {
        float temp = h_A[idx]; 
        h_A[idx] = h_B[idx];
	h_B[idx] = temp;
    } 
}


int main() {


    float *h_A, *h_B, *d_A, *d_B;
    h_A = new float[DSIZE];
    h_B = new float[DSIZE];


    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    std::cout << "******** Before Swap ********" << std::endl;
    std::cout << "h_A = ";
    for (int i = 0; i < 10; ++i) {
        std::cout << h_A[i] << " ";
    }   
    std::cout << std::endl;
    std::cout << "h_B = ";
    for (int i = 0; i < 10; ++i) {
        std::cout << h_B[i] << " ";
    }   
    std::cout << std::endl;

    // Allocate memory for host and device pointers 
    hipMalloc(&d_A, DSIZE*sizeof(float));
    hipMalloc(&d_B, DSIZE*sizeof(float));

    // Copy from host to device
    hipMemcpy(d_A, h_A, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    
    // Launch the kernel
    vector_addition <<< grid_size,block_size >>> (d_A, d_B);

    // Copy back to host 
    hipMemcpy(h_A, d_A, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, DSIZE*sizeof(float), hipMemcpyDeviceToHost);

    // Print and check some elements to make sure swapping was successfull
    std::cout << "******** After Swap ********" << std::endl;
    std::cout << "h_A = ";
    for (int i = 0; i < 10; ++i) {
        std::cout << h_A[i] << " ";
    }   
    std::cout << std::endl;
    std::cout << "h_B = ";
    for (int i = 0; i < 10; ++i) {
        std::cout << h_B[i] << " ";
    }   
    std::cout << std::endl;

    // Free the memory
    free(h_A);
    free(h_B);

    hipFree(d_A);
    hipFree(d_B);
    
    return 0;
}
