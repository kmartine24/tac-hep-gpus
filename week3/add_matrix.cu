
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>


const int DSIZE_X = 256;
const int DSIZE_Y = 256;

__global__ void add_matrix(float *A, float *B, float *C, int x_dim, int y_dim)
{
    //FIXME:
    // Express in terms of threads and blocks
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    // Add the two matrices - make sure you are not out of range
    if (idx < x_dim && idy < y_dim) {
        C[idx*y_dim+idy] = A[idx*y_dim+idy] + B[idx*y_dim+idy];
    }
}

int main()
{

    // Create and allocate memory for host and device pointers 
    float *h_A, *d_A, *h_B, *d_B, *h_C, * d_C;
    h_A = new float[DSIZE_X * DSIZE_Y];
    h_B = new float[DSIZE_X * DSIZE_Y];
    h_C = new float[DSIZE_X * DSIZE_Y];

    // Fill in the matrices
    // FIXME
    for (int i = 0; i < DSIZE_X; i++) {
        for (int j = 0; j < DSIZE_Y; j++) {
            //FIXME
	    h_A[DSIZE_Y*i + j] = rand()/(float)RAND_MAX;
	    h_B[DSIZE_Y*i + j] = rand()/(float)RAND_MAX;
	    h_C[DSIZE_Y*i + j] = 0;
        }
    }

    hipMalloc(&d_A, DSIZE_X*DSIZE_Y*sizeof(float));
    hipMalloc(&d_B, DSIZE_X*DSIZE_Y*sizeof(float));
    hipMalloc(&d_C, DSIZE_X*DSIZE_Y*sizeof(float));

    std::cout << "******** Before the Sum: ********" << std::endl;
    for (int i = 0; i < 5; ++i) {
        for (int j = 0; j < 5; ++j) {
	    std::cout << "A["<<i<<"][" << j << "] = "; 
	    std::cout << h_A[i*DSIZE_Y+j] << " ";
	}
	std::cout << std::endl;
    }
    std::cout << std::endl;
    for (int i = 0; i < 5; ++i) {
        for (int j = 0; j < 5; ++j) {
	    std::cout << "B["<<i<<"][" << j << "] = "; 
	    std::cout << h_B[i*DSIZE_Y+j] << " ";
	}
	std::cout << std::endl;
    }

    // Copy from host to device
    hipMemcpy(d_A, h_A, DSIZE_X*DSIZE_Y*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE_X*DSIZE_Y*sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    // dim3 is a built in CUDA type that allows you to define the block 
    // size and grid size in more than 1 dimentions
    // Syntax : dim3(Nx,Ny,Nz)
    dim3 blockSize(32,32); 
    dim3 gridSize(1,1); 
    
    add_matrix<<<gridSize, blockSize>>>(d_A, d_B, d_C, DSIZE_X, DSIZE_Y);

    // Copy back to host 
    hipMemcpy(h_C, d_C, DSIZE_X*DSIZE_Y*sizeof(float), hipMemcpyDeviceToHost);

    // Print and check some elements to make the addition was succesfull
    std::cout << "******** After the Sum: ********" << std::endl;
    for (int i = 0; i < 5; ++i) {
        for (int j = 0; j < 5; ++j) {
	    std::cout << "C["<<i<<"][" << j << "] = "; 
	    std::cout << h_C[i*DSIZE_Y+j] << " ";
	}
	std::cout << std::endl;
    }
    // Free the memory     
    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
